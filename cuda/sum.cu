
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <assert.h>

#define N 100000000

__global__ void add(float* c, float* a, float* b, unsigned int n) {
	unsigned int i = threadIdx.x;
	unsigned int stride = blockDim.x;

	unsigned int tid = blockIdx.x*blockDim.x + threadIdx.x;
	if (tid < n)
		c[tid] = a[tid] + b[tid];
}


int main() {
	float* a = (float*)malloc(N*sizeof(float));
	float* b = (float*)malloc(N*sizeof(float));
	float* c = (float*)malloc(N*sizeof(float));

	for (unsigned int i = 0; i < N; ++i) {
		a[i] = 1.0;
		b[i] = 2.0;
	}

	float* da;
	float* db;
	float* dc;
	hipMalloc(&da, N*sizeof(float));
	hipMalloc(&db, N*sizeof(float));
	hipMalloc(&dc, N*sizeof(float));

	hipMemcpy(da, a, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(db, b, N*sizeof(float), hipMemcpyHostToDevice);

	add<<<N/256,256>>>(dc,db,da,N);

	hipMemcpy(c, dc, N*sizeof(float), hipMemcpyDeviceToHost);

	for (unsigned int i = 0; i < N; ++i) {
		assert(c[i] == 3.0f);
	}

	return 0;
}
