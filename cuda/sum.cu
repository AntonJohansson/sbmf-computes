
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <assert.h>

#define N 100000000

__global__ void add(float* c, float* a, float* b, unsigned int n) {
	for (unsigned int i = 0; i < n; ++i) {
		c[i] = a[i] + b[i];
	}
}

int main() {
	float* a = (float*)malloc(N*sizeof(float));
	float* b = (float*)malloc(N*sizeof(float));
	float* c = (float*)malloc(N*sizeof(float));

	for (unsigned int i = 0; i < N; ++i) {
		a[i] = 1.0;
		b[i] = 2.0;
	}

	float* da;
	float* db;
	float* dc;
	hipMalloc(&da, N*sizeof(float));
	hipMalloc(&db, N*sizeof(float));
	hipMalloc(&dc, N*sizeof(float));

	hipMemcpy(da, a, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(db, b, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dc, c, N*sizeof(float), hipMemcpyHostToDevice);

	add<<<1,1>>>(dc,db,da,N);

	hipMemcpy(a, da, N*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(b, db, N*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(c, dc, N*sizeof(float), hipMemcpyDeviceToHost);

	for (unsigned int i = 0; i < N; ++i) {
		assert(c[i] == 3.0f);
	}

	return 0;
}
